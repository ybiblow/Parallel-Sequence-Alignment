#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ int strchr_CUDA(char* arr, char a){
	for(;;arr++){
		if(*arr == a)
			return 1;
		if(*arr == '\0')
			return 0;
	}
	
}

__device__ int is_identical_CUDA(char a, char b){
	if(a == b)
		return 1;
	return 0;
}

__device__ int is_conservative_CUDA(char a, char b){
	int i;
	const char* conservative_groups[9] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
	for(i = 0; i < 9; i++){
		if(strchr_CUDA((char*)conservative_groups[i], a) && strchr_CUDA((char*)conservative_groups[i], b))
			return 1;
	}
	return 0;
}

__device__ int is_semi_conservative_CUDA(char a, char b){
	int i;
	const char* semi_conservative_groups[11] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};
	for(i = 0; i < 11; i++){
		if(strchr_CUDA((char*)semi_conservative_groups[i], a) && strchr_CUDA((char*)semi_conservative_groups[i], b))
			return 1;
	}
	return 0;
}

__device__ char find_similarity_Kernel(char a, char b) {
	if(is_identical_CUDA(a, b))
		return '*';
	else if(is_conservative_CUDA(a, b))
		return ':';
	else if(is_semi_conservative_CUDA(a, b))
		return '.';
	else
		return ' ';
}

__global__ void calc_similarity_CUDA_Kernel(char *d_seq1, char* d_mutant, int d_mutant_len, char* d_result, int offset) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Increment the proper value of the arrray according to thread ID 
	if (i < d_mutant_len){
		d_result[i] = find_similarity_Kernel(d_seq1[i + offset], d_mutant[i]);
	}
}

__global__ void incrementByOne(int *arr, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Increment the proper value of the arrray according to thread ID 
    if (i < numElements)
        arr[i]++;
}

__global__ void get_Mutant_CUDA_Kernel(char *sequence, char* d_tmp_mutant, int len, int m, int n) {
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		//char* tmp_mutant = (char*)malloc((len-2)*sizeof(char));
		if (i < len){
			if(i < (m - 1))
				d_tmp_mutant[i] = sequence[i];
			else if(i > (m - 1) && i < (n - 1))
				d_tmp_mutant[i - 1] = sequence[i];
			else if(i > (n - 1))
				d_tmp_mutant[i - 2] = sequence[i];
		}
		if(i == len)
			d_tmp_mutant[i - 2] = '\0';
			
    /*
	//printf("getting mutant (%d,%d) length = %d\n", m, n, len);	
	int i;
	int f_index = m - 1;
	int e_index = n - 1;
	char* mutant = (char*)malloc((len - 1) * sizeof(char));	
	#pragma omp parallel for shared(f_index, e_index, mutant)
	for(i = 0; i < len; i++){
		if(i < f_index)
			mutant[i] = sequence[i];
		else if(i > f_index && i < e_index)
			mutant[i - 1] = sequence[i];
		else if(i > e_index)
			mutant[i - 2] = sequence[i];
	}
	mutant[len - 2] = '\0';	
	return mutant;
	*/


    // Increment the proper value of the arrray according to thread ID 
}

int computeOnGPU(int *data, int numElements) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);
  

    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    incrementByOne<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(data, d_A, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

char* get_Mutant_CUDA(char* sequence,int len, int m, int n){

	char* mutant = (char*)malloc((len - 2) * sizeof(char));
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	size_t size = len * sizeof(char);
  

	// Allocate memory on GPU to copy the data from the host
	char* d_tmp_mutant;
	err = hipMalloc((void **)&d_tmp_mutant, (len - 2) * sizeof(char));      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	// Allocate memory on GPU to copy the data from the host
	char* d_sequence;
	err = hipMalloc((void **)&d_sequence, size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy data from host to the GPU memory
	err = hipMemcpy(d_sequence, sequence, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Launch the Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(len + threadsPerBlock - 1) / threadsPerBlock;
	get_Mutant_CUDA_Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_sequence, d_tmp_mutant, len, m, n);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(mutant, d_tmp_mutant, (len - 2) * sizeof(char), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free allocated memory on GPU
	if (hipFree(d_tmp_mutant) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	// Free allocated memory on GPU
	if (hipFree(d_sequence) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	//printf("%s\n%s\n",sequence, mutant);
	
	return mutant;
}

void calc_best_score_CUDA(char* seq1, char* mutant, float* weights,int* best_offset, float* best_score){
	int i;
	int offset = strlen(seq1) - strlen(mutant);
	int tmp_score;
	*best_score = 0;
	//printf("offset = %d\n", offset);
	for(i = 0; i <= offset; i++){
		// working on changing calc_similarity_CUDA to work with cuda
		char* similarity = calc_similarity_CUDA(seq1, mutant, i);
		tmp_score = calc_score(similarity, strlen(mutant), &weights[0]);
		if(tmp_score > *best_score){
			*best_score = tmp_score;
			*best_offset = i;
		}
		free(similarity);
	}
	//printf("Best offset is: %d\n", *best_offset);
	
}

char* calc_similarity_CUDA(char* seq1, char* mutant, int offset){
	int len = strlen(mutant);
	size_t size = len * sizeof(char);
	char* result = (char*)malloc(len * sizeof(char));
	
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;	
  
	// Allocate memory on GPU to copy the data from the host
	char* d_seq1;
	err = hipMalloc((void **)&d_seq1, strlen(seq1) * sizeof(char));      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Allocate memory on GPU to copy the data from the host
	char* d_mutant;
	err = hipMalloc((void **)&d_mutant, size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Allocate memory on GPU to copy the data from the host
	char* d_result;
	err = hipMalloc((void **)&d_result, size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory
	err = hipMemcpy(d_seq1, seq1, strlen(seq1) * sizeof(char), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory
	err = hipMemcpy(d_mutant, mutant, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Launch the Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(len + threadsPerBlock - 1) / threadsPerBlock;
	calc_similarity_CUDA_Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_seq1, d_mutant, len, d_result, offset);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU
	if (hipFree(d_seq1) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU
	if (hipFree(d_mutant) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU
	if (hipFree(d_result) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	printf("offset = %d\n", offset);
	return result;
}
