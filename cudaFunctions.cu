#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ int strchr_CUDA(char* arr, char a){
	for(;;arr++){
		if(*arr == a)
			return 1;
		if(*arr == '\0')
			return 0;
	}
	
}

__device__ int is_identical_CUDA(char a, char b){
	if(a == b)
		return 1;
	return 0;
}

__device__ int is_conservative_CUDA(char a, char b){
	int i;
	const char* conservative_groups[9] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
	for(i = 0; i < 9; i++){
		if(strchr_CUDA((char*)conservative_groups[i], a) && strchr_CUDA((char*)conservative_groups[i], b))
			return 1;
	}
	return 0;
}

__device__ int is_semi_conservative_CUDA(char a, char b){
	int i;
	const char* semi_conservative_groups[11] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};
	for(i = 0; i < 11; i++){
		if(strchr_CUDA((char*)semi_conservative_groups[i], a) && strchr_CUDA((char*)semi_conservative_groups[i], b))
			return 1;
	}
	return 0;
}

__device__ char find_similarity_Kernel(char a, char b) {
	if(is_identical_CUDA(a, b))
		return '*';
	else if(is_conservative_CUDA(a, b))
		return ':';
	else if(is_semi_conservative_CUDA(a, b))
		return '.';
	else
		return ' ';
}

__global__ void calc_similarity_CUDA_Kernel(char *d_seq1, char* d_mutant, int d_mutant_len, char* d_result, int offset) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Increment the proper value of the arrray according to thread ID 
	if (i < d_mutant_len){
		d_result[i] = find_similarity_Kernel(d_seq1[i + offset], d_mutant[i]);
	}
}

__global__ void incrementByOne(int *arr, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Increment the proper value of the arrray according to thread ID 
    if (i < numElements)
        arr[i]++;
}

__device__	void calc_score_cuda(char* seq1, char* mutant, float* score, float* weights, int mutant_len){
	int i, stars = 0, colons = 0, dots = 0, spaces = 0;
	char* similarity_arr = (char*)malloc(mutant_len * sizeof(char));
	for(i = 0; i < mutant_len; i++){
		similarity_arr[i] = find_similarity_Kernel(seq1[i], mutant[i]);
		if(similarity_arr[i] == '*')
			stars++;
		else if(similarity_arr[i] == ':')
			colons++;
		else if(similarity_arr[i] == '.')
			dots++;
		else if(similarity_arr[i] == ' ')
			spaces++;
	}
	free(similarity_arr);
	*score = weights[0] * stars - weights[1] * colons - weights[2] * dots - weights[3] * spaces;
}

__global__	void calcBestScore(char* d_seq1, char* d_mutant, float* d_score_arr, float* weights, int offset, int mutant_len) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Increment the proper value of the arrray according to thread ID 
	if (i <= offset){
		calc_score_cuda(&d_seq1[i], d_mutant, &d_score_arr[i], weights, mutant_len);
	}
}

__global__ void get_Mutant_CUDA_Kernel(char *sequence, char* d_tmp_mutant, int len, int m, int n) {
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		//char* tmp_mutant = (char*)malloc((len-2)*sizeof(char));
		if (i < len){
			if(i < (m - 1))
				d_tmp_mutant[i] = sequence[i];
			else if(i > (m - 1) && i < (n - 1))
				d_tmp_mutant[i - 1] = sequence[i];
			else if(i > (n - 1))
				d_tmp_mutant[i - 2] = sequence[i];
		}
		if(i == len)
			d_tmp_mutant[i - 2] = '\0';
			
    /*
	//printf("getting mutant (%d,%d) length = %d\n", m, n, len);	
	int i;
	int f_index = m - 1;
	int e_index = n - 1;
	char* mutant = (char*)malloc((len - 1) * sizeof(char));	
	#pragma omp parallel for shared(f_index, e_index, mutant)
	for(i = 0; i < len; i++){
		if(i < f_index)
			mutant[i] = sequence[i];
		else if(i > f_index && i < e_index)
			mutant[i - 1] = sequence[i];
		else if(i > e_index)
			mutant[i - 2] = sequence[i];
	}
	mutant[len - 2] = '\0';	
	return mutant;
	*/


    // Increment the proper value of the arrray according to thread ID 
}

int computeOnGPU(int *data, int numElements) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);
  

    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    incrementByOne<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(data, d_A, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

char* get_Mutant_CUDA(char* sequence,int len, int m, int n){

	char* mutant = (char*)malloc((len - 2) * sizeof(char));
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	size_t size = len * sizeof(char);
  

	// Allocate memory on GPU to copy the data from the host
	char* d_tmp_mutant;
	err = hipMalloc((void **)&d_tmp_mutant, (len - 2) * sizeof(char));      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	// Allocate memory on GPU to copy the data from the host
	char* d_sequence;
	err = hipMalloc((void **)&d_sequence, size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy data from host to the GPU memory
	err = hipMemcpy(d_sequence, sequence, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Launch the Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(len + threadsPerBlock - 1) / threadsPerBlock;
	get_Mutant_CUDA_Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_sequence, d_tmp_mutant, len, m, n);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(mutant, d_tmp_mutant, (len - 2) * sizeof(char), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free allocated memory on GPU
	if (hipFree(d_tmp_mutant) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	// Free allocated memory on GPU
	if (hipFree(d_sequence) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	//printf("%s\n%s\n",sequence, mutant);
	
	return mutant;
}

void calc_best_score_CUDA(char* seq1, char* mutant, float* weights,int* best_offset, float* best_score){
	int i;
	int offset = strlen(seq1) - strlen(mutant);
	int tmp_score;
	*best_score = 0;
	//printf("offset = %d\n", offset);
	for(i = 0; i <= offset; i++){
		// working on changing calc_similarity_CUDA to work with cuda
		char* similarity = calc_similarity_CUDA(seq1, mutant, i);
		tmp_score = calc_score(similarity, strlen(mutant), &weights[0]);
		if(tmp_score > *best_score){
			*best_score = tmp_score;
			*best_offset = i;
		}
		free(similarity);
	}
	//printf("Best offset is: %d\n", *best_offset);
	
}

void calc_best_score_CUDA_1(char* seq1, char* mutant, float* weights, int* best_offset, float* best_score){
	int seq1_len = strlen(seq1);
	size_t seq1_size = seq1_len * sizeof(char);
	int mutant_len = strlen(mutant);
	size_t mutant_size = mutant_len * sizeof(char);
	int offset = seq1_len - mutant_len;
	size_t d_score_size = offset * sizeof(float);
	size_t d_weights_size = 4 * sizeof(float);
	
	float* score_arr = (float*)malloc(offset * sizeof(float));
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
	
	// Allocate memory on GPU to copy the data from the host - d_seq1
	char* d_seq1;
	err = hipMalloc((void **)&d_seq1, seq1_size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Allocate memory on GPU to copy the data from the host - d_mutant
	char* d_mutant;
	err = hipMalloc((void **)&d_mutant, mutant_size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Allocate memory on GPU to copy the data from the host - d_score_arr
	float* d_score_arr;
	err = hipMalloc((void **)&d_score_arr, d_score_size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Allocate memory on GPU to copy the data from the host - d_weights
	float* d_weights;
	err = hipMalloc((void **)&d_weights, d_weights_size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory - d_seq1
	err = hipMemcpy(d_seq1, seq1, seq1_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory - d_mutant
	err = hipMemcpy(d_mutant, mutant, mutant_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory - d_weights
	err = hipMemcpy(d_weights, weights, d_weights_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	//void calcBestScore(char* d_seq1, char* d_mutant, int* d_score_arr, float* weights, int offset, int mutant_len)	

	// Launch the Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(offset + threadsPerBlock - 1) / threadsPerBlock;
	calcBestScore<<<blocksPerGrid, threadsPerBlock>>>(d_seq1, d_mutant, d_score_arr, d_weights, offset, mutant_len);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy the  result from GPU to the host memory. d_score_arr ---> score_arr
	err = hipMemcpy(score_arr, d_score_arr, d_score_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// score_arr has the info we need to get the best score
	int score = -1;
	*best_offset = -1;
	int i;
	for(i = 0; i < offset; i++){
		if(score_arr[i] > score){
			score = score_arr[i];
			*best_offset = i;
		}
	}
	*best_offset = offset;
	*best_score = score;
	
	// Free allocated memory on GPU - d_seq1
	if (hipFree(d_seq1) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU - d_mutant
	if (hipFree(d_mutant) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	// Free allocated memory on GPU - d_score_arr
	if (hipFree(d_score_arr) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU - d_weights
	if (hipFree(d_weights) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	free(score_arr);
	//*best_score = ;
	//*best_offset = ;
	//printf("I'm here, This is where you want to be!!!\n");
}
char* calc_similarity_CUDA(char* seq1, char* mutant, int offset){
	int len = strlen(mutant);
	size_t size = len * sizeof(char);
	char* result = (char*)malloc(len * sizeof(char));
	
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;	
  
	// Allocate memory on GPU to copy the data from the host
	char* d_seq1;
	err = hipMalloc((void **)&d_seq1, strlen(seq1) * sizeof(char));      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Allocate memory on GPU to copy the data from the host
	char* d_mutant;
	err = hipMalloc((void **)&d_mutant, size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Allocate memory on GPU to copy the data from the host
	char* d_result;
	err = hipMalloc((void **)&d_result, size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory
	err = hipMemcpy(d_seq1, seq1, strlen(seq1) * sizeof(char), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory
	err = hipMemcpy(d_mutant, mutant, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Launch the Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(len + threadsPerBlock - 1) / threadsPerBlock;
	calc_similarity_CUDA_Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_seq1, d_mutant, len, d_result, offset);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU
	if (hipFree(d_seq1) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU
	if (hipFree(d_mutant) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	// Free allocated memory on GPU
	if (hipFree(d_result) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	printf("offset = %d\n", offset);
	return result;
}
