#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

#define CUDA_ERR_CHECK(err,msg) {if (err != hipSuccess) {fprintf(stderr, msg " - %s\n", hipGetErrorString(err));exit(EXIT_FAILURE);}}

#define CUDA_MEM_INIT(d_pointer, size, type) {hipError_t err = hipSuccess;\
					size_t arrSize = size * sizeof(type);\
				err = hipMalloc((void**)&d_pointer, arrSize);\
			CUDA_ERR_CHECK(err, "Failed to allocate device memory");}
	
#define CUDA_MEM_INIT_COPY(dest, src, size, type) {\
			hipError_t err = hipSuccess;\
		size_t  arrSize = size * sizeof(type);\
		err = hipMalloc((void**)&dest, arrSize);\
CUDA_ERR_CHECK(err, "Failed to allocate device memory");\
err = hipMemcpy(dest, src, arrSize, hipMemcpyHostToDevice);\
CUDA_ERR_CHECK(err, "Failed to copy data from host to device"); }

__device__ int strchr_CUDA(char* arr, char a){
	for(;;arr++){
		if(*arr == a)
			return 1;
		if(*arr == '\0')
			return 0;
	}
	
}

__device__ int is_identical_CUDA(char a, char b){
	if(a == b)
		return 1;
	return 0;
}

__device__ int is_conservative_CUDA(char a, char b){
	int i;
	const char* conservative_groups[9] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
	for(i = 0; i < 9; i++){
		if(strchr_CUDA((char*)conservative_groups[i], a) && strchr_CUDA((char*)conservative_groups[i], b))
			return 1;
	}
	return 0;
}

__device__ int is_semi_conservative_CUDA(char a, char b){
	int i;
	const char* semi_conservative_groups[11] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};
	for(i = 0; i < 11; i++){
		if(strchr_CUDA((char*)semi_conservative_groups[i], a) && strchr_CUDA((char*)semi_conservative_groups[i], b))
			return 1;
	}
	return 0;
}

__device__ char find_similarity_Kernel(char a, char b) {
	if(is_identical_CUDA(a, b))
		return '*';
	else if(is_conservative_CUDA(a, b))
		return ':';
	else if(is_semi_conservative_CUDA(a, b))
		return '.';
	else
		return ' ';
}

__global__ void calc_similarity_CUDA_Kernel(char *d_seq1, char* d_mutant, int d_mutant_len, char* d_result, int offset) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Increment the proper value of the arrray according to thread ID 
	if (i < d_mutant_len){
		d_result[i] = find_similarity_Kernel(d_seq1[i + offset], d_mutant[i]);
	}
}

__device__	void calc_score_cuda(char* seq1, char* mutant, float* score, float* weights, int mutant_len){
	int i, stars = 0, colons = 0, dots = 0, spaces = 0;
	char* similarity_arr = (char*)malloc(mutant_len * sizeof(char));
	for(i = 0; i < mutant_len; i++){
		similarity_arr[i] = find_similarity_Kernel(seq1[i], mutant[i]);
		if(similarity_arr[i] == '*')
			stars++;
		else if(similarity_arr[i] == ':')
			colons++;
		else if(similarity_arr[i] == '.')
			dots++;
		else if(similarity_arr[i] == ' ')
			spaces++;
	}
	free(similarity_arr);
	*score = weights[0] * stars - weights[1] * colons - weights[2] * dots - weights[3] * spaces;
}

__global__	void calcBestScore(char* d_seq1, char* d_mutant, float* d_score_arr, float* weights, int offset, int mutant_len) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Increment the proper value of the arrray according to thread ID 
	if (i <= offset){
		calc_score_cuda(&d_seq1[i], d_mutant, &d_score_arr[i], weights, mutant_len);
	}
}

__global__ void get_Mutant_CUDA_Kernel(char *sequence, char* d_tmp_mutant, int len, int m, int n) {
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		//char* tmp_mutant = (char*)malloc((len-2)*sizeof(char));
		if (i < len){
			if(i < (m - 1))
				d_tmp_mutant[i] = sequence[i];
			else if(i > (m - 1) && i < (n - 1))
				d_tmp_mutant[i - 1] = sequence[i];
			else if(i > (n - 1))
				d_tmp_mutant[i - 2] = sequence[i];
		}
		if(i == len)
			d_tmp_mutant[i - 2] = '\0';
}

char* get_Mutant_CUDA(char* sequence,int len, int m, int n){

	char* mutant = (char*)malloc((len - 2) * sizeof(char));
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	size_t size = len * sizeof(char);
  

	// Allocate memory on GPU to copy the data from the host
	char* d_tmp_mutant;
	err = hipMalloc((void **)&d_tmp_mutant, (len - 2) * sizeof(char));      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	// Allocate memory on GPU to copy the data from the host
	char* d_sequence;
	err = hipMalloc((void **)&d_sequence, size);      
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy data from host to the GPU memory
	err = hipMemcpy(d_sequence, sequence, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Launch the Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(len + threadsPerBlock - 1) / threadsPerBlock;
	get_Mutant_CUDA_Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_sequence, d_tmp_mutant, len, m, n);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(mutant, d_tmp_mutant, (len - 2) * sizeof(char), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free allocated memory on GPU
	if (hipFree(d_tmp_mutant) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	// Free allocated memory on GPU
	if (hipFree(d_sequence) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	//printf("%s\n%s\n",sequence, mutant);
	
	return mutant;
}

__device__ void CUDAGetNK(int mutant_num, int seq2_len, int* n, int* k)
{
	int i;
	int num_of_mutants_in_row = seq2_len;

	for(i = 1; i < seq2_len; i++){
		if(mutant_num - (num_of_mutants_in_row - 1) > 0){
		    mutant_num -= (num_of_mutants_in_row - 1);
		    num_of_mutants_in_row--;
		}else{
		    break;
		}
	}
	
	*n = i;	
	*k = i + mutant_num;
}

//	returns the score for a given mutant and offset, the func will compare 2 chars each loop, one from seq1 and another from seq2, it will skip the letters in indexes N and K,
//	that will make the comparison of the two strings a comparison between seq2 and a mutant sequence(N,K)
__device__ float calcMutantScore(char* seq1, char* seq2, float* d_conservative_matrix,int len2, int n, int k, int index, int offset)
{
	// N and K needs to be minus 1 because for thread i=0 we will get (N,K)=(1,2) and we don't want to skip 1,2 we want to skip indexes (0,1) therefore we subtract 1 from each
	n = n - 1;
	k = k - 1;
	float score = 0;
	int i = 0, j = i;
	for (i = 0; i < len2 - 2; i++, j++)
	{
		if(j == n)
			j++; // skipping the comparison with index n
		if(j == k)
			j++; // skipping the comparison with index k
		float tmp_score = d_conservative_matrix[(seq1[i] - 'A') * 26 + (seq2[j] - 'A')]; // the comparison matrix is used here as a 2d array but in fact it is a 1d array
		score += tmp_score;
	}	
	//printf("(%d,%d) %1.2f\n",n, k, score);
	return score;	
}

//	each thread is responsible for a single mutant and calculates its best score and offset, example:
//	thread i=0 will calculate the result for (N=1,K=2) and store the best score it could find in d_mutantsBestScores[0] and best offset it could find in d_mutantsBestOffsets[0]
__global__ void calcMutantBestScoreKernel(char* d_seq1, char* d_seq2, float* d_comp_matrix, float* d_mutantsBestScores, int* d_mutantsBestOffsets, int num_mutants, int maxOffset, int len2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n,k;
	float bestScore = -10000;
	int offset = 0;
	if (i < num_mutants)
	{
		CUDAGetNK(i+1, len2, &n, &k);
		//printf("i = %d, (%d,%d)\n", i, n, k);
		for (int j = 0; j < maxOffset; j++)
		{
			float score = calcMutantScore(&d_seq1[j], d_seq2, d_comp_matrix, len2, n, k, i, j);
			// asking if the score we found for the given (N,K) and offset is bigger than the best score we found so far
			if (score > bestScore)
			{
				bestScore = score;
				offset = j;	
			}
		}
		// storing the final results in the 2 vectors
		d_mutantsBestScores[i]	= bestScore;
		d_mutantsBestOffsets[i] = offset;
	}
}

/*
	this function calculates the best scores and offsets for all the mutants, the result of those results will be in two vectors: mutantsBestScores, mutantsBestOffsets. the
	length of those 2 vectors is the number of mutants for a given seq2.
	this function uses CUDA to calculate the score and offset for each mutant and stores the results in the previously mentioned vectors.
	later it uses OMP to get the best score, best offset and (N,K) for the given seq2.
	return the string result that is to be printed.
*/
char* calc_best_score_CUDA(char* seq1, char* seq2, float* comp_matrix){
	
	int seq1_len = strlen(seq1);
	int seq2_len = strlen(seq2);
	
	//printf("seq1_len = %d, seq2_len = %d\n", seq1_len, seq2_len);
	//printf("seq2 = %s\n", seq2);
	
	// calc maxmimum offset and number of mutants
	int maxOffset = seq1_len - (seq2_len - 2) + 1;
	int num_of_mutants = seq2_len * (seq2_len - 1) / 2;
	
	// allocate memory in CPU for 2 vectors
	float* mutantsBestScores = (float*) malloc(num_of_mutants * sizeof(float));
	int* mutantsBestOffsets = (int*) malloc(num_of_mutants * sizeof(int));
	
	// allocate d_seq1, d_seq2, d_comp_matrix memory and copy data to device
	char* d_seq1 = NULL;
	char* d_seq2 = NULL;
	float* d_comp_matrix = NULL;
	CUDA_MEM_INIT_COPY(d_seq1, seq1, seq1_len, char);
	CUDA_MEM_INIT_COPY(d_seq2, seq2, seq2_len, char);
	CUDA_MEM_INIT_COPY(d_comp_matrix, comp_matrix, SIZE_OF_COMP_MATRIX, float);
	
	// allocate memory for d_mutantsBestScores & d_mutantsBestOffsets and dont copy data! - the data will be filled in the device
	float* d_mutantsBestScores = NULL; 
	int* d_mutantsBestOffsets = NULL;
	CUDA_MEM_INIT(d_mutantsBestScores, num_of_mutants, float);
	CUDA_MEM_INIT(d_mutantsBestOffsets, num_of_mutants, int);
	
	int threads = 256;
	int blocks = (num_of_mutants + threads - 1) / threads;
	
	calcMutantBestScoreKernel<<<blocks, threads>>>(d_seq1, d_seq2, d_comp_matrix, d_mutantsBestScores, d_mutantsBestOffsets, num_of_mutants, maxOffset, seq2_len);
	
	hipMemcpy(mutantsBestScores, d_mutantsBestScores, num_of_mutants * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(mutantsBestOffsets, d_mutantsBestOffsets, num_of_mutants * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i < num_of_mutants; i++){
		//printf("i = %d, score = %1.2f\n", i, mutantsBestScores[i]);
	}
	/*
	float maxScore = -10000;
	int bestOffset = 0;
	int bestMutantNum = -1;
	
	for (int i = 0; i < num_of_mutants; i++)
	{
		if (mutantsBestScores[i] > maxScore)
		{
			maxScore = mutantsBestScores[i];
			bestOffset = mutantsBestOffsets[i];
			bestMutantNum = i;
		}
	}
	int n,k;
	CPUGetNK(bestMutantNum + 1, seq2_len, &n, &k);
	printf("mutant num: %d, MS(%d,%d), score: %1.2f, offset: %d\n", bestMutantNum, n, k, maxScore, bestOffset);
	*/
	
	// using OMP to get the best results	
	char* final_result = calcBestScoreOmp(mutantsBestScores, mutantsBestOffsets, num_of_mutants, seq2_len);
	printf("%s", final_result);
	return final_result;
}
